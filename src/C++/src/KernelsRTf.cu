#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <iterator>
#include <cmath>
#include <array>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#include "GUtils.h"
#include "Structs.h"
#include "InterfaceReflector.h"
//#include "CompOverload.h"

#define CSIZERT 5

/* Kernels for single precision PO.
 * Author: Arend Moerman
 * For questions, contact: arendmoerman@gmail.com
 */

// Declare constant memory for Device
__constant__ float conrt[CSIZERT]; // a, b, c, t0, epsilon
__constant__ float mat[16]; //
__constant__ int nTot;
__constant__ int cflip;

/**
 * Wrapper for finding errors in CUDA API calls.
 *
 * @param code The errorcode returned from failed API call.
 * @param file The file in which failure occured.
 * @param line The line in file in which error occured.
 * @param abort Exit code upon error.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__ void _debugArrayf(float arr[3])
{
    printf("%f, %f, %f\n", arr[0], arr[1], arr[2]);
}

__device__ __inline__ float common1(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*(xr + t*dxr)/(conrt[0]*conrt[0]) + (yr + t*dyr)*(yr + t*dyr)/(conrt[1]*conrt[1]);
}


__device__ __inline__ float common2(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*2*dxr/(conrt[0]*conrt[0]) + (yr + t*dyr)*2*dyr/(conrt[1]*conrt[1]);
}


__device__ __inline__ float gp(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - common1(t, xr, yr, dxr, dyr)) /
                (dzr - common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float gh(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - conrt[2]*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) /
                (dzr - conrt[2]/(2*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) *
                common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float ge(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - conrt[2]*sqrt(1 - common1(t, xr, yr, dxr, dyr))) /
                (dzr + conrt[2]/(2*sqrt(1 - common1(t, xr, yr, dxr, dyr))) *
                common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float gpl(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr) / dzr;
}


__device__ __inline__ void np(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = -2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


__device__ __inline__ void nh(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = -2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = 2 * zr / (conrt[2]*conrt[2]) * cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}

__device__ __inline__ void ne(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = 2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = 2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = 2 * zr / (conrt[2]*conrt[2]) * cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


__device__ __inline__ void npl(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = 0;
    out[1] = 0;
    out[2] = cflip;
}

// Not placed in GUtils.h, because want to place rotMat in constant memory
__device__ __inline__ void matVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2];
        }
    }

    else
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2] + mat[3+n*4];
        }
    }
}

__device__ __inline__ void invmatVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2];
        }
    }

    else
    {
        float temp;
        for(int n=0; n<3; n++)
        {
            temp = -mat[n]*mat[3] - mat[n+4]*mat[7] - mat[n+8]*mat[11];
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2] + temp;
        }
    }
}

__device__ __inline__ void transfRays(float *x, float *y, float *z,
                                  float *dx, float *dy, float *dz,
                                  int i, bool inv = false)
{
    bool vec = true;
    float inp[3], out[3];

    //if (i == 300) {printf("%f\n", mat[0]);}

    inp[0] = x[i];
    inp[1] = y[i];
    inp[2] = z[i];

    if (inv) {invmatVec4(inp, out);}
    else {matVec4(inp, out);}

    x[i] = out[0];
    y[i] = out[1];
    z[i] = out[2];

    inp[0] = dx[i];
    inp[1] = dy[i];
    inp[2] = dz[i];

    if (inv) {invmatVec4(inp, out, vec);}
    else {matVec4(inp, out, vec);}

    dx[i] = out[0];
    dy[i] = out[1];
    dz[i] = out[2];

}

__host__ std::array<dim3, 2> _initCUDA(reflparamsf ctp, float epsilon, float t0,
                                      int _nTot, int nBlocks, int nThreads)
{
    // Calculate nr of blocks per grid and nr of threads per block
    dim3 nrb(nBlocks); dim3 nrt(nThreads);

    // Pack constant array
    hipFloatComplex _conrt[CSIZERT] = {ctp.coeffs[0], ctp.coeffs[1],
                                  ctp.coeffs[2], t0, epsilon};

    int iflip = 1;
    if (ctp.flip) {iflip = -1;}

    // Copy constant array to Device constant memory
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(conrt), &_conrt, CSIZERT * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(mat), ctp.transf, 16 * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(nTot), &_nTot, sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cflip), &iflip, sizeof(int)) );

    std::array<dim3, 2> BT;
    BT[0] = nrb;
    BT[1] = nrt;

    return BT;
}

__global__ void propagateRaysToP(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gp(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }

        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        np(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToH(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gh(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        nh(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToE(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = ge(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        ne(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToPl(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx < nTot)
    {
      //if (idx == 0) {printf("%f\n", conrt[2]);}
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];
        //printf("%f\n", x);
        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gpl(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        //printf("%.14f\n", check);

        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        npl(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

extern "C" void callRTKernel(reflparamsf ctp, cframef *fr_in,
                            cframef *fr_out, float epsilon, float t0,
                            int nBlocks, int nThreads)
{
    std::array<dim3, 2> BT;
    BT = _initCUDA(ctp, epsilon, t0, fr_in->size, nBlocks, nThreads);

    float *d_xs, *d_ys, *d_zs;
    gpuErrchk( hipMalloc((void**)&d_xs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_ys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_xs, fr_in->x, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_ys, fr_in->y, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_zs, fr_in->z, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_dxs, *d_dys, *d_dzs;
    gpuErrchk( hipMalloc((void**)&d_dxs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_dxs, fr_in->dx, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dys, fr_in->dy, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dzs, fr_in->dz, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_xt, *d_yt, *d_zt;
    gpuErrchk( hipMalloc((void**)&d_xt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_yt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zt, fr_in->size * sizeof(float)) );

    float *d_dxt, *d_dyt, *d_dzt;
    gpuErrchk( hipMalloc((void**)&d_dxt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dyt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzt, fr_in->size * sizeof(float)) );

    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;

    printf("Calculating ray-trace...\n");
    begin = std::chrono::steady_clock::now();

    if (ctp.type == 0)
    {
        propagateRaysToP<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        //printf("made it\n");
        gpuErrchk( hipDeviceSynchronize() );

    }

    else if (ctp.type == 1)
    {
        propagateRaysToH<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    else if (ctp.type == 2)
    {
        propagateRaysToE<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    else if (ctp.type == 3)
    {
        propagateRaysToPl<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time : "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()
              << " [us]\n" << std::endl;

    gpuErrchk( hipMemcpy(fr_out->x, d_xt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->y, d_yt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->z, d_zt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(fr_out->dx, d_dxt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dy, d_dyt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dz, d_dzt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
}
