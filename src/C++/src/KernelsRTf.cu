#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <iterator>
#include <cmath>
#include <array>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#include "GUtils.h"
#include "Structs.h"
#include "InterfaceReflector.h"
//#include "CompOverload.h"

#define CSIZE 5
#define MILLISECOND 1000

/* Kernels for single precision PO.
 * Author: Arend Moerman
 * For questions, contact: arendmoerman@gmail.com
 */

// Declare constant memory for Device
__constant__ float con[CSIZE]; // a, b, c, t0, epsilon
__constant__ float mat[16]; //
__constant__ int nTot;
__constant__ int cflip;

/**
 * Wrapper for finding errors in CUDA API calls.
 *
 * @param code The errorcode returned from failed API call.
 * @param file The file in which failure occured.
 * @param line The line in file in which error occured.
 * @param abort Exit code upon error.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__ void _debugArrayf(float arr[3])
{
    printf("%f, %f, %f\n", arr[0], arr[1], arr[2]);
}

__device__ __inline__ float common1(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*(xr + t*dxr)/(con[0]*con[0]) + (yr + t*dyr)*(yr + t*dyr)/(con[1]*con[1]);
}


__device__ __inline__ float common2(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*2*dxr/(con[0]*con[0]) + (yr + t*dyr)*2*dyr/(con[1]*con[1]);
}


__device__ __inline__ float gp(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - common1(t, xr, yr, dxr, dyr)) /
                (dzr - common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float gh(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - con[2]*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) /
                (dzr - con[2]/(2*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) *
                common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float ge(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - con[2]*sqrt(1 - common1(t, xr, yr, dxr, dyr))) /
                (dzr + con[2]/(2*sqrt(1 - common1(t, xr, yr, dxr, dyr))) *
                common2(t, xr, yr, dxr, dyr));
}


__device__ __inline__ float gpl(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr) / dzr;
}


__device__ __inline__ void np(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (con[0]*con[0]) * cflip;
    out[1] = -2 * yr / (con[1]*con[1]) * cflip;
    out[2] = cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


__device__ __inline__ void nhe(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (con[0]*con[0]) * cflip;
    out[1] = -2 * yr / (con[1]*con[1]) * cflip;
    out[2] = 2 * zr / (con[2]*con[2]) * cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


__device__ __inline__ void npl(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = 0;
    out[1] = 0;
    out[2] = cflip;
}

// Not placed in GUtils.h, because want to place rotMat in constant memory
__device__ __inline__ void matVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2];
        }
    }

    else
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2] + mat[3+n*4];
        }
    }
}

__device__ __inline__ void invmatVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2];
        }
    }

    else
    {
        float temp;
        for(int n=0; n<3; n++)
        {
            temp = -mat[n]*mat[3] - mat[n+4]*mat[7] - mat[n+8]*mat[11];
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2] + temp;
        }
    }
}

__device__ __inline__ void transfRays(float *x, float *y, float *z,
                                  float *dx, float *dy, float *dz,
                                  int i, bool inv = false)
{
    bool vec = true;
    float inp[3], out[3];

    inp[0] = x[i];
    inp[1] = y[i];
    inp[2] = z[i];

    if (inv) {invmatVec4(inp, out);}
    else {matVec4(inp, out);}

    x[i] = out[0];
    y[i] = out[1];
    z[i] = out[2];

    inp[0] = dx[i];
    inp[1] = dy[i];
    inp[2] = dz[i];

    if (inv) {invmatVec4(inp, out, vec);}
    else {matVec4(inp, out, vec);}

    dx[i] = out[0];
    dy[i] = out[1];
    dz[i] = out[2];

}

__host__ std::array<dim3, 2> _initCUDA(reflparamsf ctp, float epsilon, float t0,
                                      int nTot, int nBlocks, int nThreads)
{
    // Calculate nr of blocks per grid and nr of threads per block
    dim3 nrb(nBlocks); dim3 nrt(nThreads);

    // Pack constant array
    hipFloatComplex _con[CSIZE] = {ctp.coeffs[0], ctp.coeffs[1],
                                  ctp.coeffs[2], nTot, t0, epsilon};

    int iflip = 1;
    if (ctp.flip) {iflip = -1;}

    // Copy constant array to Device constant memory
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(con), &_con, CSIZE * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(mat), &ctp.transf, 16 * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cflip), &iflip, sizeof(int)) );

    std::array<dim3, 2> BT;
    BT[0] = nrb;
    BT[1] = nrt;

    return BT;
}

__global__ void propagateRaysToP(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = con[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > con[4])
        {
            t1 = gp(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        np(xt[idx], yt[idx], zt[idx], norms);
        check = (dxt[idx]*norms[0] + dyt[idx]*norms[1] + dzt[idx]*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToH(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = con[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > con[4])
        {
            t1 = gh(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        nhe(xt[idx], yt[idx], zt[idx], norms);
        check = (dxt[idx]*norms[0] + dyt[idx]*norms[1] + dzt[idx]*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToE(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = con[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > con[4])
        {
            t1 = ge(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        nhe(xt[idx], yt[idx], zt[idx], norms);
        check = (dxt[idx]*norms[0] + dyt[idx]*norms[1] + dzt[idx]*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

__global__ void propagateRaysToPl(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = con[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > con[4])
        {
            t1 = gpl(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        npl(xt[idx], yt[idx], zt[idx], norms);
        check = (dxt[idx]*norms[0] + dyt[idx]*norms[1] + dzt[idx]*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

extern "C" void callRTKernel(reflparamsf ctp, cframef *fr_in,
                            cframef *fr_out, float epsilon, float t0,
                            int nBlocks, int nThreads)
{
    std::array<dim3, 2> BT;
    BT = _initCUDA(ctp, epsilon, t0, fr_in->size, nBlocks, nThreads);

    float *d_xs, *d_ys, *d_zs;
    gpuErrchk( hipMalloc((void**)&d_xs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_ys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_xs, fr_in->x, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_ys, fr_in->y, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_zs, fr_in->z, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_dxs, *d_dys, *d_dzs;
    gpuErrchk( hipMalloc((void**)&d_dxs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_dxs, fr_in->dx, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dys, fr_in->dy, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dzs, fr_in->dz, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_xt, *d_yt, *d_zt;
    gpuErrchk( hipMalloc((void**)&d_xt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_yt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zt, fr_in->size * sizeof(float)) );

    float *d_dxt, *d_dyt, *d_dzt;
    gpuErrchk( hipMalloc((void**)&d_dxt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dyt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzt, fr_in->size * sizeof(float)) );

    if (ctp.type == 0)
    {
        propagateRaysToP<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
    }

    else if (ctp.type == 1)
    {
        propagateRaysToH<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
    }

    else if (ctp.type == 2)
    {
        propagateRaysToE<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
    }

    else if (ctp.type == 3)
    {
        propagateRaysToPl<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
    }

    gpuErrchk( hipMemcpy(fr_out->x, d_xt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->y, d_yt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->z, d_zt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(fr_out->dx, d_dxt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dy, d_dyt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dz, d_dzt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
}
