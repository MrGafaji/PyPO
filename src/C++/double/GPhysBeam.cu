#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <fstream>
#include <string>
#include <iterator>
#include <array>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "GDataHandler.h"
#include "GUtils.h"

#define M_PI            3.14159265358979323846  /* pi */
#define C_L             2.99792458e11 // mm s^-1
#define MU_0            1.2566370614e-3 // kg mm s^-2 A^-2
#define EPS_VAC         1 / (MU_0 * C_L*C_L)
#define ZETA_0_INV      1 / (C_L * MU_0)

#define CSIZE 10

/* This program calculates the PO propagation between a source and a target plane.
 * NOTE: This file contains the CUDA version of PhysBeam
 * 
 * In order to run, the presence of the following .txt files in the POPPy/src/C++/input/ is required:
 * - s_Jr_(x,y,z).txt the real x,y,z components of the source electric current distribution
 * - s_Ji_(x,y,z).txt the imaginary x,y,z components of the source electric current distribution 
 * - s_Mr_(x,y,z).txt the real x,y,z components of the source magnetic current distribution
 * - s_Mi_(x,y,z).txt the imaginary x,y,z components of the source magnetic current distribution
 *
 * - s_(x,y,z).txt the source x,y,z grids
 * - s_n(x,y,z).txt the source nx,ny,nz normal grids
 * - A_s the source area elements corresponding to points x,y,z
 * 
 * - t_(x,y,z).txt the target x,y,z grids
 * - t_n(x,y,z).txt the target nx,ny,nz normal grids
 * 
 * 
 * Author: Arend Moerman
 * For questions, contact: arendmoerman@gmail.com
 */

// Declare constant memory for Device
__constant__ hipDoubleComplex con[CSIZE];     // Contains: k, eps, mu0, zeta0, pi, C_l, Time direction, unit, zero, c4 as complex numbers
__constant__ double eye[3][3];      // Identity matrix
__constant__ int g_s;               // Gridsize on source
__constant__ int g_t;               // Gridsize on target

/**
 * Wrapper for finding errors in CUDA API calls.
 * 
 * @param code The errorcode returned from failed API call.
 * @param file The file in which failure occured.
 * @param line The line in file in which error occured.
 * @param abort Exit code upon error.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * Function to calculate complex exponential with
 * CUDA type hipDoubleComplex.
 * 
 * @param z Complex number.
 */
__device__ __inline__ hipDoubleComplex my_cexp(hipDoubleComplex z)
{
    hipDoubleComplex res;
    double t = exp(z.x);
    double ys = sin(z.y);
    double yc = cos(z.y);
    res = hipCmul(make_hipDoubleComplex(t, 0.), make_hipDoubleComplex(yc, ys));
    return res;
}

/**
 * Calculate total field at point on target.
 * 
 * @param d_xs C-style array containing source points x-coordinate.
 * @param d_ys C-style array containing source points y-coordinate.
 * @param d_zs C-style array containing source points z-coordinate.
 * @param d_Jx C-style array containing source J x-component.
 * @param d_Jy C-style array containing source J y-component.
 * @param d_Jz C-style array containing source J z-component.
 * @param d_Mx C-style array containing source M x-component.
 * @param d_My C-style array containing source M y-component.
 * @param d_Mz C-style array containing source M z-component.
 * @param point C-style array of length 3 containing xyz coordinates of target point.
 * @param d_A C-style array containing area elements.
 * @param d_ei C-style array of length 3 to be filled with E-field at point.
 * @param d_hi C-style array of length 3 to be filled with H-field at point.
 */ 
__device__ void fieldAtPoint(double *d_xs, double *d_ys, double*d_zs, 
                    hipDoubleComplex *d_Jx, hipDoubleComplex *d_Jy, hipDoubleComplex *d_Jz, 
                    hipDoubleComplex *d_Mx, hipDoubleComplex *d_My, hipDoubleComplex *d_Mz, 
                    double (&point)[3], double *d_A, 
                    hipDoubleComplex (&d_ei)[3], hipDoubleComplex (&d_hi)[3])
{
    // Scalars (double & complex double)
    double r;                           // Distance between source and target points
    double r_inv;                       // 1 / r
    hipDoubleComplex omega;                       // Angular frequency of field
    hipDoubleComplex Green;         // Container for Green's function
    hipDoubleComplex r_in_s;        // Container for inner products between wavevctor and currents
    hipDoubleComplex rc;
    
    // Arrays of doubles
    double source_point[3]; // Container for xyz co-ordinates
    double r_vec[3];        // Distance vector between source and target points
    double k_hat[3];        // Unit wavevctor
    double k_arr[3];        // Wavevector
    
    // Arrays of complex doubles
    hipDoubleComplex e_field[3] = {con[8], con[8], con[8]}; // Electric field on target
    hipDoubleComplex h_field[3] = {con[8], con[8], con[8]}; // Magnetic field on target
    hipDoubleComplex js[3];             // Electric current at source point
    hipDoubleComplex ms[3];             // Magnetic current at source point
    hipDoubleComplex e_vec_thing[3];    // Electric current contribution to e-field
    hipDoubleComplex h_vec_thing[3];    // Magnetic current contribution to h-field
    hipDoubleComplex k_out_ms[3];       // Outer product between k and ms
    hipDoubleComplex k_out_js[3];       // Outer product between k and js
    hipDoubleComplex temp[3];           // Temporary container for intermediate values

    //e_field = {con[8], con[8], con[8]};
    //h_field = {con[8], con[8], con[8]};
    
    omega = hipCmul(con[5], con[0]); // C_L * k
    
    for(int i=0; i<g_s; i++)
    {
        js[0] = d_Jx[i];
        js[1] = d_Jy[i];
        js[2] = d_Jz[i];
        
        ms[0] = d_Mx[i];
        ms[1] = d_My[i];
        ms[2] = d_Mz[i];
        
        source_point[0] = d_xs[i];
        source_point[1] = d_ys[i];
        source_point[2] = d_zs[i];
        
        diff(point, source_point, r_vec);
        abs(r_vec, r);                              
        
        rc = make_hipDoubleComplex(r, 0.);        
        r_inv = 1 / r;
        
        s_mult(r_vec, r_inv, k_hat);
        s_mult(k_hat, con[0].x, k_arr);
        
        // e-field
        dot(k_hat, js, r_in_s);
        s_mult(k_hat, r_in_s, temp);
        diff(js, temp, e_vec_thing);
        
        ext(k_arr, ms, k_out_ms);
        
        // h-field
        dot(k_hat, ms, r_in_s);
        s_mult(k_hat, r_in_s, temp);
        diff(ms, temp, h_vec_thing);
        
        ext(k_arr, js, k_out_js);
        
        hipDoubleComplex d_Ac = make_hipDoubleComplex(d_A[i], 0.);
        
        Green = hipCmul(hipCdiv(my_cexp(hipCmul(con[6], hipCmul(con[7], hipCmul(con[0], rc)))), (hipCmul(con[9], hipCmul(con[4], rc)))), hipCmul(d_Ac, con[7]));

        for( int n=0; n<3; n++)
        {
            e_field[n] = hipCsub(e_field[n], hipCmul(hipCsub(hipCmul(omega, hipCmul(con[2], e_vec_thing[n])), k_out_ms[n]), Green));
            h_field[n] = hipCsub(h_field[n], hipCmul(hipCadd(hipCmul(omega, hipCmul(con[1], h_vec_thing[n])), k_out_js[n]), Green));
        }  
    }

    d_ei[0] = e_field[0];
    d_ei[1] = e_field[1];
    d_ei[2] = e_field[2];
    
    d_hi[0] = h_field[0];
    d_hi[1] = h_field[1];
    d_hi[2] = h_field[2];

    
}

/**
 * Kernel for toPrint == 0: save J and M.
 * 
 * @param d_xs C-style array containing source points x-coordinate.
 * @param d_ys C-style array containing source points y-coordinate.
 * @param d_zs C-style array containing source points z-coordinate.
 * @param d_A C-style array containing area elements.
 * @param d_xt C-style array containing target points x-coordinate.
 * @param d_yt C-style array containing target points y-coordinate.
 * @param d_zt C-style array containing target points z-coordinate.
 * @param d_nxt C-style array containing target norms x-component.
 * @param d_nyt C-style array containing target norms y-component.
 * @param d_nzt C-style array containing target norms z-component.
 * @param d_Jx C-style array containing source J x-component.
 * @param d_Jy C-style array containing source J y-component.
 * @param d_Jz C-style array containing source J z-component.
 * @param d_Mx C-style array containing source M x-component.
 * @param d_My C-style array containing source M y-component.
 * @param d_Mz C-style array containing source M z-component.
 * @param d_Jxt C-style array to be filled with target J x-component.
 * @param d_Jyt C-style array to be filled with target J y-component.
 * @param d_Jzt C-style array to be filled with target J z-component.
 * @param d_Mxt C-style array to be filled with target M x-component.
 * @param d_Myt C-style array to be filled with target M y-component.
 * @param d_Mzt C-style array to be filled with target M z-component.
 */ 
__global__ void GpropagateBeam_0(double *d_xs, double *d_ys, double *d_zs,
                                double *d_A, double *d_xt, double *d_yt, double *d_zt,
                                double *d_nxt, double *d_nyt, double *d_nzt,
                                hipDoubleComplex *d_Jx, hipDoubleComplex *d_Jy, hipDoubleComplex *d_Jz,
                                hipDoubleComplex *d_Mx, hipDoubleComplex *d_My, hipDoubleComplex *d_Mz,
                                hipDoubleComplex *d_Jxt, hipDoubleComplex *d_Jyt, hipDoubleComplex *d_Jzt,
                                hipDoubleComplex *d_Mxt, hipDoubleComplex *d_Myt, hipDoubleComplex *d_Mzt)
{
    
    // Scalars (double & complex double)
    hipDoubleComplex e_dot_p_r_perp;    // E-field - perpendicular reflected POI polarization vector dot product
    hipDoubleComplex e_dot_p_r_parr;    // E-field - parallel reflected POI polarization vector dot product
    
    // Arrays of doubles
    double S_i_norm[3];         // Normalized incoming Poynting vector
    double p_i_perp[3];         // Perpendicular incoming POI polarization vector 
    double p_i_parr[3];         // Parallel incoming POI polarization vector 
    double S_r_norm[3];         // Normalized reflected Poynting vector
    double p_r_perp[3];         // Perpendicular reflected POI polarization vector 
    double p_r_parr[3];         // Parallel reflected POI polarization vector 
    double S_out_n[3];          // Container for Poynting-normal ext products
    double point[3];            // Point on target
    double norms[3];            // Normal vector at point
    double e_out_h_r[3];        // Real part of E-field - H-field ext product

    // Arrays of complex doubles
    hipDoubleComplex e_r[3];            // Reflected E-field
    hipDoubleComplex h_r[3];            // Reflected H-field
    hipDoubleComplex n_out_e_i_r[3];    // Electric current
    hipDoubleComplex temp1[3];          // Temporary container 1 for intermediate irrelevant values
    hipDoubleComplex temp2[3];          // Temporary container 2
    hipDoubleComplex temp3[3];          // Temporary container 3
    
    // Return containers
    hipDoubleComplex d_ei[3];
    hipDoubleComplex d_hi[3];

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < g_t)
    {
        point[0] = d_xt[idx];
        point[1] = d_yt[idx];
        point[2] = d_zt[idx];

        norms[0] = d_nxt[idx];
        norms[1] = d_nyt[idx];
        norms[2] = d_nzt[idx];

        // Calculate total incoming E and H field at point on target
        fieldAtPoint(d_xs, d_ys, d_zs, 
                    d_Jx, d_Jy, d_Jz, 
                    d_Mx, d_My, d_Mz, 
                    point, d_A, d_ei, d_hi);

        // Calculate normalised incoming poynting vector.
        conja(d_ei, temp1);                        // h_conj
        ext(d_hi, temp1, temp2);                  // e_out_h
        
        for (int n=0; n<3; n++) 
        {
            e_out_h_r[n] = hipCreal(temp2[n]);                      // e_out_h_r
        }

        normalize(e_out_h_r, S_i_norm);                       // S_i_norm                   
        
        // Calculate incoming polarization vectors
        ext(S_i_norm, norms, S_out_n);                      // S_i_out_n
        normalize(S_out_n, p_i_perp);                       // p_i_perp                   
        ext(p_i_perp, S_i_norm, p_i_parr);               // p_i_parr                     
        
        // Now calculate reflected poynting vector.
        snell(S_i_norm, norms, S_r_norm);                // S_r_norm     

        // Calculate normalised reflected polarization vectors
        ext(S_r_norm, norms, S_out_n);                      // S_r_out_n
        normalize(S_out_n, p_r_perp);                       // p_r_perp                   
        ext(S_r_norm, p_r_perp, p_r_parr);               // p_r_parr                     
        
        // Now, calculate reflected field from target
        dot(d_ei, p_r_perp, e_dot_p_r_perp);      // e_dot_p_r_perp
        dot(d_ei, p_r_parr, e_dot_p_r_parr);      // e_dot_p_r_parr

        // Calculate reflected field from reflection matrix
        for(int n=0; n<3; n++)
        {
            e_r[n] = hipCsub(hipCmul(e_dot_p_r_perp, make_hipDoubleComplex(-p_i_perp[n], 0.)), hipCmul(e_dot_p_r_parr, make_hipDoubleComplex(p_i_parr[n], 0.)));
        }

        ext(S_r_norm, e_r, temp1);                       // h_r_temp
        s_mult(temp1, con[3], h_r);                     // ZETA_0_INV, h_r   

        //Calculate and store J and M only
        for(int n=0; n<3; n++)
        {
            temp1[n] = hipCadd(e_r[n], d_ei[n]); // e_i_r
            temp2[n] = hipCadd(h_r[n], d_hi[n]); // h_i_r
        } 
            
        ext(norms, temp2, temp3);
        
        d_Jxt[idx] = temp3[0];
        d_Jyt[idx] = temp3[1];
        d_Jzt[idx] = temp3[2];
        
        ext(norms, temp1, n_out_e_i_r);
        s_mult(n_out_e_i_r, -1., temp3);
        
        d_Mxt[idx] = temp3[0];
        d_Myt[idx] = temp3[1];
        d_Mzt[idx] = temp3[2];
    }
}

/**
 * Kernel for toPrint == 1: save Ei and Hi.
 * 
 * @param d_xs C-style array containing source points x-coordinate.
 * @param d_ys C-style array containing source points y-coordinate.
 * @param d_zs C-style array containing source points z-coordinate.
 * @param d_A C-style array containing area elements.
 * @param d_xt C-style array containing target points x-coordinate.
 * @param d_yt C-style array containing target points y-coordinate.
 * @param d_zt C-style array containing target points z-coordinate.
 * @param d_Jx C-style array containing source J x-component.
 * @param d_Jy C-style array containing source J y-component.
 * @param d_Jz C-style array containing source J z-component.
 * @param d_Mx C-style array containing source M x-component.
 * @param d_My C-style array containing source M y-component.
 * @param d_Mz C-style array containing source M z-component.
 * @param d_Ext C-style array to be filled with target Ei x-component.
 * @param d_Eyt C-style array to be filled with target Ei y-component.
 * @param d_Ezt C-style array to be filled with target Ei z-component.
 * @param d_Hxt C-style array to be filled with target Hi x-component.
 * @param d_Hyt C-style array to be filled with target Hi y-component.
 * @param d_Hzt C-style array to be filled with target Hi z-component.
 */ 
__global__ void GpropagateBeam_1(double *d_xs, double *d_ys, double *d_zs,
                                double *d_A, double *d_xt, double *d_yt, double *d_zt,
                                hipDoubleComplex *d_Jx, hipDoubleComplex *d_Jy, hipDoubleComplex *d_Jz,
                                hipDoubleComplex *d_Mx, hipDoubleComplex *d_My, hipDoubleComplex *d_Mz,
                                hipDoubleComplex *d_Ext, hipDoubleComplex *d_Eyt, hipDoubleComplex *d_Ezt,
                                hipDoubleComplex *d_Hxt, hipDoubleComplex *d_Hyt, hipDoubleComplex *d_Hzt)
{
    // Arrays of doubles
    double point[3];            // Point on target
    
    // Return containers for call to fieldAtPoint
    hipDoubleComplex d_ei[3];
    hipDoubleComplex d_hi[3];

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < g_t)
    {
        point[0] = d_xt[idx];
        point[1] = d_yt[idx];
        point[2] = d_zt[idx];

        // Calculate total incoming E and H field at point on target
        fieldAtPoint(d_xs, d_ys, d_zs, 
                    d_Jx, d_Jy, d_Jz, 
                    d_Mx, d_My, d_Mz, 
                    point, d_A, d_ei, d_hi);
        
        d_Ext[idx] = d_ei[0];
        d_Eyt[idx] = d_ei[1];
        d_Ezt[idx] = d_ei[2];

        d_Hxt[idx] = d_hi[0];
        d_Hyt[idx] = d_hi[1];
        d_Hzt[idx] = d_hi[2];
    }
}

/**
 * Kernel for toPrint == 2: save J, M, Ei and Hi.
 * 
 * @param d_xs C-style array containing source points x-coordinate.
 * @param d_ys C-style array containing source points y-coordinate.
 * @param d_zs C-style array containing source points z-coordinate.
 * @param d_A C-style array containing area elements.
 * @param d_xt C-style array containing target points x-coordinate.
 * @param d_yt C-style array containing target points y-coordinate.
 * @param d_zt C-style array containing target points z-coordinate.
 * @param d_nxt C-style array containing target norms x-component.
 * @param d_nyt C-style array containing target norms y-component.
 * @param d_nzt C-style array containing target norms z-component.
 * @param d_Jx C-style array containing source J x-component.
 * @param d_Jy C-style array containing source J y-component.
 * @param d_Jz C-style array containing source J z-component.
 * @param d_Mx C-style array containing source M x-component.
 * @param d_My C-style array containing source M y-component.
 * @param d_Mz C-style array containing source M z-component.
 * @param d_Jxt C-style array to be filled with target J x-component.
 * @param d_Jyt C-style array to be filled with target J y-component.
 * @param d_Jzt C-style array to be filled with target J z-component.
 * @param d_Mxt C-style array to be filled with target M x-component.
 * @param d_Myt C-style array to be filled with target M y-component.
 * @param d_Mzt C-style array to be filled with target M z-component.
 * @param d_Ext C-style array to be filled with target Ei x-component.
 * @param d_Eyt C-style array to be filled with target Ei y-component.
 * @param d_Ezt C-style array to be filled with target Ei z-component.
 * @param d_Hxt C-style array to be filled with target Hi x-component.
 * @param d_Hyt C-style array to be filled with target Hi y-component.
 * @param d_Hzt C-style array to be filled with target Hi z-component.
 */ 
__global__ void GpropagateBeam_2(double *d_xs, double *d_ys, double *d_zs,
                                double *d_A, double *d_xt, double *d_yt, double *d_zt,
                                double *d_nxt, double *d_nyt, double *d_nzt,
                                hipDoubleComplex *d_Jx, hipDoubleComplex *d_Jy, hipDoubleComplex *d_Jz,
                                hipDoubleComplex *d_Mx, hipDoubleComplex *d_My, hipDoubleComplex *d_Mz,
                                hipDoubleComplex *d_Jxt, hipDoubleComplex *d_Jyt, hipDoubleComplex *d_Jzt,
                                hipDoubleComplex *d_Mxt, hipDoubleComplex *d_Myt, hipDoubleComplex *d_Mzt,
                                hipDoubleComplex *d_Ext, hipDoubleComplex *d_Eyt, hipDoubleComplex *d_Ezt,
                                hipDoubleComplex *d_Hxt, hipDoubleComplex *d_Hyt, hipDoubleComplex *d_Hzt)
{
    
    // Scalars (double & complex double)
    hipDoubleComplex e_dot_p_r_perp;    // E-field - perpendicular reflected POI polarization vector dot product
    hipDoubleComplex e_dot_p_r_parr;    // E-field - parallel reflected POI polarization vector dot product
    
    // Arrays of doubles
    double S_i_norm[3];         // Normalized incoming Poynting vector
    double p_i_perp[3];         // Perpendicular incoming POI polarization vector 
    double p_i_parr[3];         // Parallel incoming POI polarization vector 
    double S_r_norm[3];         // Normalized reflected Poynting vector
    double p_r_perp[3];         // Perpendicular reflected POI polarization vector 
    double p_r_parr[3];         // Parallel reflected POI polarization vector 
    double S_out_n[3];          // Container for Poynting-normal ext products
    double point[3];            // Point on target
    double norms[3];            // Normal vector at point
    double e_out_h_r[3];        // Real part of E-field - H-field ext product

    // Arrays of complex doubles
    hipDoubleComplex e_r[3];            // Reflected E-field
    hipDoubleComplex h_r[3];            // Reflected H-field
    hipDoubleComplex n_out_e_i_r[3];    // Electric current
    hipDoubleComplex temp1[3];          // Temporary container 1 for intermediate irrelevant values
    hipDoubleComplex temp2[3];          // Temporary container 2
    hipDoubleComplex temp3[3];          // Temporary container 3
    
    // Return containers
    hipDoubleComplex d_ei[3];
    hipDoubleComplex d_hi[3];

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < g_t)
    {
        point[0] = d_xt[idx];
        point[1] = d_yt[idx];
        point[2] = d_zt[idx];

        norms[0] = d_nxt[idx];
        norms[1] = d_nyt[idx];
        norms[2] = d_nzt[idx];

        // Calculate total incoming E and H field at point on target
        fieldAtPoint(d_xs, d_ys, d_zs, 
                    d_Jx, d_Jy, d_Jz, 
                    d_Mx, d_My, d_Mz, 
                    point, d_A, d_ei, d_hi);
        
        d_Ext[idx] = d_ei[0];
        d_Eyt[idx] = d_ei[1];
        d_Ezt[idx] = d_ei[2];

        d_Hxt[idx] = d_hi[0];
        d_Hyt[idx] = d_hi[1];
        d_Hzt[idx] = d_hi[2];

        // Calculate normalised incoming poynting vector.
        conja(d_ei, temp1);                        // h_conj
        ext(d_hi, temp1, temp2);                  // e_out_h
        
        for (int n=0; n<3; n++) 
        {
            e_out_h_r[n] = hipCreal(temp2[n]);                      // e_out_h_r
        }

        normalize(e_out_h_r, S_i_norm);                       // S_i_norm                   
        
        // Calculate incoming polarization vectors
        ext(S_i_norm, norms, S_out_n);                      // S_i_out_n
        normalize(S_out_n, p_i_perp);                       // p_i_perp                   
        ext(p_i_perp, S_i_norm, p_i_parr);               // p_i_parr                     
        
        // Now calculate reflected poynting vector.
        snell(S_i_norm, norms, S_r_norm);                // S_r_norm     

        // Calculate normalised reflected polarization vectors
        ext(S_r_norm, norms, S_out_n);                      // S_r_out_n
        normalize(S_out_n, p_r_perp);                       // p_r_perp                   
        ext(S_r_norm, p_r_perp, p_r_parr);               // p_r_parr                     
        
        // Now, calculate reflected field from target
        dot(d_ei, p_r_perp, e_dot_p_r_perp);      // e_dot_p_r_perp
        dot(d_ei, p_r_parr, e_dot_p_r_parr);      // e_dot_p_r_parr

        // Calculate reflected field from reflection matrix
        for(int n=0; n<3; n++)
        {
            e_r[n] = hipCsub(hipCmul(e_dot_p_r_perp, make_hipDoubleComplex(-p_i_perp[n], 0.)), hipCmul(e_dot_p_r_parr, make_hipDoubleComplex(p_i_parr[n], 0.)));
        }

        ext(S_r_norm, e_r, temp1);                       // h_r_temp
        s_mult(temp1, con[3], h_r);                     // ZETA_0_INV, h_r   

        //Calculate and store J and M only
        for(int n=0; n<3; n++)
        {
            temp1[n] = hipCadd(e_r[n], d_ei[n]); // e_i_r
            temp2[n] = hipCadd(h_r[n], d_hi[n]); // h_i_r
        } 
            
        ext(norms, temp2, temp3);
        
        d_Jxt[idx] = temp3[0];
        d_Jyt[idx] = temp3[1];
        d_Jzt[idx] = temp3[2];
        
        ext(norms, temp1, n_out_e_i_r);
        s_mult(n_out_e_i_r, -1., temp3);
        
        d_Mxt[idx] = temp3[0];
        d_Myt[idx] = temp3[1];
        d_Mzt[idx] = temp3[2];
    }
}

/**
 * Kernel for toPrint == 3: save Pr, Er and Hr.
 * 
 * @param d_xs C-style array containing source points x-coordinate.
 * @param d_ys C-style array containing source points y-coordinate.
 * @param d_zs C-style array containing source points z-coordinate.
 * @param d_A C-style array containing area elements.
 * @param d_xt C-style array containing target points x-coordinate.
 * @param d_yt C-style array containing target points y-coordinate.
 * @param d_zt C-style array containing target points z-coordinate.
 * @param d_nxt C-style array containing target norms x-component.
 * @param d_nyt C-style array containing target norms y-component.
 * @param d_nzt C-style array containing target norms z-component.
 * @param d_Jx C-style array containing source J x-component.
 * @param d_Jy C-style array containing source J y-component.
 * @param d_Jz C-style array containing source J z-component.
 * @param d_Mx C-style array containing source M x-component.
 * @param d_My C-style array containing source M y-component.
 * @param d_Mz C-style array containing source M z-component.
 * @param d_Prxt C-style array to be filled with Pr x-component.
 * @param d_Pryt C-style array to be filled with Pr y-component.
 * @param d_Przt C-style array to be filled with Pr z-component.
 * @param d_Ext C-style array to be filled with target Er x-component.
 * @param d_Eyt C-style array to be filled with target Er y-component.
 * @param d_Ezt C-style array to be filled with target Er z-component.
 * @param d_Hxt C-style array to be filled with target Hr x-component.
 * @param d_Hyt C-style array to be filled with target Hr y-component.
 * @param d_Hzt C-style array to be filled with target Hr z-component.
 */ 
__global__ void GpropagateBeam_3(double *d_xs, double *d_ys, double *d_zs,
                                double *d_A, double *d_xt, double *d_yt, double *d_zt,
                                double *d_nxt, double *d_nyt, double *d_nzt,
                                hipDoubleComplex *d_Jx, hipDoubleComplex *d_Jy, hipDoubleComplex *d_Jz,
                                hipDoubleComplex *d_Mx, hipDoubleComplex *d_My, hipDoubleComplex *d_Mz,
                                double *d_Prxt, double *d_Pryt, double *d_Przt,
                                hipDoubleComplex *d_Ext, hipDoubleComplex *d_Eyt, hipDoubleComplex *d_Ezt,
                                hipDoubleComplex *d_Hxt, hipDoubleComplex *d_Hyt, hipDoubleComplex *d_Hzt)
{
    
    // Scalars (double & complex double)
    hipDoubleComplex e_dot_p_r_perp;    // E-field - perpendicular reflected POI polarization vector dot product
    hipDoubleComplex e_dot_p_r_parr;    // E-field - parallel reflected POI polarization vector dot product
    
    // Arrays of doubles
    double S_i_norm[3];         // Normalized incoming Poynting vector
    double p_i_perp[3];         // Perpendicular incoming POI polarization vector 
    double p_i_parr[3];         // Parallel incoming POI polarization vector 
    double S_r_norm[3];         // Normalized reflected Poynting vector
    double p_r_perp[3];         // Perpendicular reflected POI polarization vector 
    double p_r_parr[3];         // Parallel reflected POI polarization vector 
    double S_out_n[3];          // Container for Poynting-normal ext products
    double point[3];            // Point on target
    double norms[3];            // Normal vector at point
    double e_out_h_r[3];        // Real part of E-field - H-field ext product

    // Arrays of complex doubles
    hipDoubleComplex e_r[3];            // Reflected E-field
    hipDoubleComplex h_r[3];            // Reflected H-field
    hipDoubleComplex temp1[3];          // Temporary container 1 for intermediate irrelevant values
    hipDoubleComplex temp2[3];          // Temporary container 2
    
    // Return containers
    hipDoubleComplex d_ei[3];
    hipDoubleComplex d_hi[3];

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < g_t)
    {
        point[0] = d_xt[idx];
        point[1] = d_yt[idx];
        point[2] = d_zt[idx];

        norms[0] = d_nxt[idx];
        norms[1] = d_nyt[idx];
        norms[2] = d_nzt[idx];

        // Calculate total incoming E and H field at point on target
        fieldAtPoint(d_xs, d_ys, d_zs, 
                    d_Jx, d_Jy, d_Jz, 
                    d_Mx, d_My, d_Mz, 
                    point, d_A, d_ei, d_hi);

        // Calculate normalised incoming poynting vector.
        conja(d_ei, temp1);                        // h_conj
        ext(d_hi, temp1, temp2);                  // e_out_h
        
        for (int n=0; n<3; n++) 
        {
            e_out_h_r[n] = hipCreal(temp2[n]);                      // e_out_h_r
        }

        normalize(e_out_h_r, S_i_norm);                       // S_i_norm                   
        
        // Calculate incoming polarization vectors
        ext(S_i_norm, norms, S_out_n);                      // S_i_out_n
        normalize(S_out_n, p_i_perp);                       // p_i_perp                   
        ext(p_i_perp, S_i_norm, p_i_parr);               // p_i_parr                     
        
        // Now calculate reflected poynting vector.
        snell(S_i_norm, norms, S_r_norm);                // S_r_norm   
        
        // Store REFLECTED Pynting vectors
        d_Prxt[idx] = S_r_norm[0];
        d_Pryt[idx] = S_r_norm[1];
        d_Przt[idx] = S_r_norm[2];

        // Calculate normalised reflected polarization vectors
        ext(S_r_norm, norms, S_out_n);                      // S_r_out_n
        normalize(S_out_n, p_r_perp);                       // p_r_perp                   
        ext(S_r_norm, p_r_perp, p_r_parr);               // p_r_parr                     
        
        // Now, calculate reflected field from target
        dot(d_ei, p_r_perp, e_dot_p_r_perp);      // e_dot_p_r_perp
        dot(d_ei, p_r_parr, e_dot_p_r_parr);      // e_dot_p_r_parr

        // Calculate reflected field from reflection matrix
        for(int n=0; n<3; n++)
        {
            e_r[n] = hipCsub(hipCmul(e_dot_p_r_perp, make_hipDoubleComplex(-p_i_perp[n], 0.)), hipCmul(e_dot_p_r_parr, make_hipDoubleComplex(p_i_parr[n], 0.)));
        }

        ext(S_r_norm, e_r, temp1);                       // h_r_temp
        s_mult(temp1, con[3], h_r);                     // ZETA_0_INV, h_r   
        
        // Store REFLECTED fields
        d_Ext[idx] = e_r[0];
        d_Eyt[idx] = e_r[1];
        d_Ezt[idx] = e_r[2];

        d_Hxt[idx] = h_r[0];
        d_Hyt[idx] = h_r[1];
        d_Hzt[idx] = h_r[2];
    }
}

int main(int argc, char *argv [])
{
    int numThreads  = atoi(argv[1]); // Number of GPU threads per block
    int numBlocks   = atoi(argv[2]); // Number of execution blocks
    double k        = atof(argv[3]); // Wavenumber of field to be propagated
    int toPrint     = atoi(argv[4]); // 0 for printing J and M, 1 for E and H and 2 for all fields
    
    double epsilon  = atof(argv[5]); // Relative electric permeability
    int prop_mode   = atoi(argv[6]); // Whether to propagate to surface or to far-field
    double t_direction = atof(argv[7]); // Whether to propagate forward or back in time
    
    int gridsize_s  = atoi(argv[8]); // Source gridsize, flattened
    int gridsize_t  = atoi(argv[9]); // Target gridsize, flattened
    
    // Calculate nr of blocks per grid and nr of threads per block
    dim3 nrb(numBlocks); dim3 nrt(numThreads);
    
    // Calculate permittivity of target
    double EPS = EPS_VAC * epsilon;
    
    // Fill ID matrix
    double _eye[3][3];
    _eye[0][0] = 1.;
    _eye[1][1] = 1.;
    _eye[2][2] = 1.;
    
    _eye[0][1] = 0.;
    _eye[0][2] = 0.;
    _eye[1][0] = 0.;
    _eye[1][2] = 0.;
    _eye[2][0] = 0.;
    _eye[2][1] = 0.;
    
    // Pack constant array
    hipDoubleComplex _con[CSIZE] = {make_hipDoubleComplex(k, 0.), 
                                    make_hipDoubleComplex(EPS, 0.), 
                                    make_hipDoubleComplex(MU_0, 0.), 
                                    make_hipDoubleComplex(ZETA_0_INV, 0.), 
                                    make_hipDoubleComplex(M_PI, 0.), 
                                    make_hipDoubleComplex(C_L, 0.),
                                    make_hipDoubleComplex(t_direction, 0.),
                                    make_hipDoubleComplex(0., 1.),
                                    make_hipDoubleComplex(0., 0.),
                                    make_hipDoubleComplex(4., 0.)};

    // Copy constant array to Device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(con), &_con, CSIZE * sizeof(hipDoubleComplex));
    hipMemcpyToSymbol(HIP_SYMBOL(eye), &_eye, sizeof(_eye));
    hipMemcpyToSymbol(HIP_SYMBOL(g_s), &gridsize_s, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(g_t), &gridsize_t, sizeof(int));

    std::string source = "s"; 
    std::string target = "t"; 
    
    GDataHandler ghandler;
    
    double source_area[gridsize_s];
    
    // Obtain source area elements
    ghandler.cppToCUDA_area(source_area);
    
    std::array<double*, 3> grid_source = ghandler.cppToCUDA_3DGrid(source);
    std::array<double*, 3> grid_target3D;
    std::array<double*, 2> grid_target2D;
    std::array<double*, 3> norm_target;
    
    // Allocate source grid and area on Device
    double *d_xs; hipMalloc( (void**)&d_xs, gridsize_s * sizeof(double) );
    double *d_ys; hipMalloc( (void**)&d_ys, gridsize_s * sizeof(double) );
    double *d_zs; hipMalloc( (void**)&d_zs, gridsize_s * sizeof(double) );
    double *d_A; hipMalloc( (void**)&d_A, gridsize_s * sizeof(double) );
    
    // Copy data from Host to Device
    hipMemcpy(d_xs, grid_source[0], gridsize_s * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, grid_source[1], gridsize_s * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_zs, grid_source[2], gridsize_s * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_A, source_area, gridsize_s * sizeof(double), hipMemcpyHostToDevice);
    
    // Declare pointers to Device arrays. No return for kernel calls
    double *d_xt; double *d_yt; double *d_zt; double *d_nxt; double *d_nyt; double *d_nzt;
    
    if (prop_mode == 0)
    {
        // Convert .txt files to CUDA arrays
        grid_target3D = ghandler.cppToCUDA_3DGrid(target);
        norm_target = ghandler.cppToCUDA_3Dnormals();
        
        // Allocate memory on Device for 3D grids and normals
        hipMalloc( (void**)&d_xt, gridsize_t * sizeof(double) );
        hipMalloc( (void**)&d_yt, gridsize_t * sizeof(double) );
        hipMalloc( (void**)&d_zt, gridsize_t * sizeof(double) );
        
        hipMalloc( (void**)&d_nxt, gridsize_t * sizeof(double) );
        hipMalloc( (void**)&d_nyt, gridsize_t * sizeof(double) );
        hipMalloc( (void**)&d_nzt, gridsize_t * sizeof(double) );
        
        // Copy grids and normals from Host to Device
        hipMemcpy(d_xt, grid_target3D[0], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_yt, grid_target3D[1], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_zt, grid_target3D[2], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        
        hipMemcpy(d_nxt, norm_target[0], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_nyt, norm_target[1], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_nzt, norm_target[2], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
    }
    
    else if (prop_mode == 1)
    {
        // Convert .txt files to CUDA arrays
        grid_target2D = ghandler.cppToCUDA_2DGrid();
        
        // Allocate memory on Device for 2D grids
        hipMalloc( (void**)&d_xt, gridsize_t * sizeof(double) );
        hipMalloc( (void**)&d_yt, gridsize_t * sizeof(double) );
        
        // Copy to GPU from Host
        hipMemcpy(d_xt, grid_target2D[0], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_yt, grid_target2D[1], gridsize_t * sizeof(double), hipMemcpyHostToDevice);
    }
    
    // Read source currents from .txt and convert to CUDA array
    std::array<hipDoubleComplex*, 3> Js = ghandler.cppToCUDA_Js();
    std::array<hipDoubleComplex*, 3> Ms = ghandler.cppToCUDA_Ms();

    // Allocate memory on Device for source currents
    hipDoubleComplex *d_Jx; hipMalloc( (void**)&d_Jx, gridsize_s * sizeof(hipDoubleComplex) );
    hipDoubleComplex *d_Jy; hipMalloc( (void**)&d_Jy, gridsize_s * sizeof(hipDoubleComplex) );
    hipDoubleComplex *d_Jz; hipMalloc( (void**)&d_Jz, gridsize_s * sizeof(hipDoubleComplex) );
    
    hipDoubleComplex *d_Mx; hipMalloc( (void**)&d_Mx, gridsize_s * sizeof(hipDoubleComplex) );
    hipDoubleComplex *d_My; hipMalloc( (void**)&d_My, gridsize_s * sizeof(hipDoubleComplex) );
    hipDoubleComplex *d_Mz; hipMalloc( (void**)&d_Mz, gridsize_s * sizeof(hipDoubleComplex) );
    
    // Copy source currents from Host to Device
    hipMemcpy(d_Jx, Js[0], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_Jy, Js[1], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_Jz, Js[2], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        
    hipMemcpy(d_Mx, Ms[0], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_My, Ms[1], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_Mz, Ms[2], gridsize_s * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    
    
    // Create Device arrays for storing the data and call the kernel
    // Which kernel is called is determined by toPrint
    if (toPrint == 0)
    {
        // Allocate memory for J and M arrays on Device
        hipDoubleComplex *d_Jxt; hipMalloc( (void**)&d_Jxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Jyt; hipMalloc( (void**)&d_Jyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Jzt; hipMalloc( (void**)&d_Jzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Mxt; hipMalloc( (void**)&d_Mxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Myt; hipMalloc( (void**)&d_Myt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Mzt; hipMalloc( (void**)&d_Mzt, gridsize_t * sizeof(hipDoubleComplex) );

        // Call to KERNEL 0
        GpropagateBeam_0<<<nrb, nrt>>>(d_xs, d_ys, d_zs, 
                                   d_A, d_xt, d_yt, d_zt,
                                   d_nxt, d_nyt, d_nzt,
                                   d_Jx, d_Jy, d_Jz,
                                   d_Mx, d_My, d_Mz,
                                   d_Jxt, d_Jyt, d_Jzt,
                                   d_Mxt, d_Myt, d_Mzt);
        hipDeviceSynchronize();
        
        // Allocate, on stackframe, Host arrays for J and M
        hipDoubleComplex h_Jxt[gridsize_t];
        hipDoubleComplex h_Jyt[gridsize_t];
        hipDoubleComplex h_Jzt[gridsize_t];
        
        hipDoubleComplex h_Mxt[gridsize_t];
        hipDoubleComplex h_Myt[gridsize_t];
        hipDoubleComplex h_Mzt[gridsize_t];
        
        // Copy content of Device J,M arrays into Host J,M arrays
        hipMemcpy(h_Jxt, d_Jxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Jyt, d_Jyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Jzt, d_Jzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Mxt, d_Mxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Myt, d_Myt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Mzt, d_Mzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // Free Device memory
        hipDeviceReset();
        
        // Pack CUDA arrays into cpp array for processing
        std::array<hipDoubleComplex*, 3> CJ;
        std::array<hipDoubleComplex*, 3> CM;
        
        // Fill the C++ std::array with C-style arrays
        CJ[0] = h_Jxt;
        CJ[1] = h_Jyt;
        CJ[2] = h_Jzt;
        
        CM[0] = h_Mxt;
        CM[1] = h_Myt;
        CM[2] = h_Mzt;
        
        // Convert the CUDA style arrays to format compatible with CPU functions
        std::vector<std::array<std::complex<double>, 3>> Jt = ghandler.CUDAToCpp_C(CJ, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Mt = ghandler.CUDAToCpp_C(CM, gridsize_t);

        std::string Jt_file = "Jt";
        std::string Mt_file = "Mt";
        
        // Write using standard CPU DataHandler object
        ghandler.dh.writeOutC(Jt, Jt_file);
        ghandler.dh.writeOutC(Mt, Mt_file);
    }

    else if (toPrint == 1)
    {
        // Allocate memory for E and H arrays on Device
        hipDoubleComplex *d_Ext; hipMalloc( (void**)&d_Ext, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Eyt; hipMalloc( (void**)&d_Eyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Ezt; hipMalloc( (void**)&d_Ezt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Hxt; hipMalloc( (void**)&d_Hxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hyt; hipMalloc( (void**)&d_Hyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hzt; hipMalloc( (void**)&d_Hzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        // Call to KERNEL 1
        GpropagateBeam_1<<<nrb, nrt>>>(d_xs, d_ys, d_zs, 
                                   d_A, d_xt, d_yt, d_zt,
                                   d_Jx, d_Jy, d_Jz,
                                   d_Mx, d_My, d_Mz,
                                   d_Ext, d_Eyt, d_Ezt,
                                   d_Hxt, d_Hyt, d_Hzt);
        hipDeviceSynchronize();
        
        // Allocate, on stackframe, Host arrays for E and H
        hipDoubleComplex h_Ext[gridsize_t];
        hipDoubleComplex h_Eyt[gridsize_t];
        hipDoubleComplex h_Ezt[gridsize_t];
        
        hipDoubleComplex h_Hxt[gridsize_t];
        hipDoubleComplex h_Hyt[gridsize_t];
        hipDoubleComplex h_Hzt[gridsize_t];
        
        // Copy content of Device E,H arrays into Host E,H arrays
        hipMemcpy(h_Ext, d_Ext, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Eyt, d_Eyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Ezt, d_Ezt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Hxt, d_Hxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hyt, d_Hyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hzt, d_Hzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // Free Device memory
        hipDeviceReset();
        
        // Pack CUDA arrays into cpp array for processing
        std::array<hipDoubleComplex*, 3> CE;
        std::array<hipDoubleComplex*, 3> CH;
        
        // Fill the C++ std::array with C-style arrays
        CE[0] = h_Ext;
        CE[1] = h_Eyt;
        CE[2] = h_Ezt;
        
        CH[0] = h_Hxt;
        CH[1] = h_Hyt;
        CH[2] = h_Hzt;
        
        // Convert the CUDA style arrays to format compatible with CPU functions
        std::vector<std::array<std::complex<double>, 3>> Et = ghandler.CUDAToCpp_C(CE, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Ht = ghandler.CUDAToCpp_C(CH, gridsize_t);

        std::string Et_file = "Et";
        std::string Ht_file = "Ht";
        
        // Write using standard CPU DataHandler object
        ghandler.dh.writeOutC(Et, Et_file);
        ghandler.dh.writeOutC(Ht, Ht_file);
    }
    
    else if (toPrint == 2)
    {
        // Allocate memory for J, M, E and H arrays on Device
        hipDoubleComplex *d_Jxt; hipMalloc( (void**)&d_Jxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Jyt; hipMalloc( (void**)&d_Jyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Jzt; hipMalloc( (void**)&d_Jzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Mxt; hipMalloc( (void**)&d_Mxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Myt; hipMalloc( (void**)&d_Myt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Mzt; hipMalloc( (void**)&d_Mzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Ext; hipMalloc( (void**)&d_Ext, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Eyt; hipMalloc( (void**)&d_Eyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Ezt; hipMalloc( (void**)&d_Ezt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Hxt; hipMalloc( (void**)&d_Hxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hyt; hipMalloc( (void**)&d_Hyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hzt; hipMalloc( (void**)&d_Hzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        // Call to KERNEL 2
        GpropagateBeam_2<<<nrb, nrt>>>(d_xs, d_ys, d_zs, 
                                   d_A, d_xt, d_yt, d_zt,
                                   d_nxt, d_nyt, d_nzt,
                                   d_Jx, d_Jy, d_Jz,
                                   d_Mx, d_My, d_Mz,
                                   d_Jxt, d_Jyt, d_Jzt,
                                   d_Mxt, d_Myt, d_Mzt,
                                   d_Ext, d_Eyt, d_Ezt,
                                   d_Hxt, d_Hyt, d_Hzt);
        hipDeviceSynchronize();
        
        // Allocate, on stackframe, Host arrays for J, M, E and H
        hipDoubleComplex h_Jxt[gridsize_t];
        hipDoubleComplex h_Jyt[gridsize_t];
        hipDoubleComplex h_Jzt[gridsize_t];
        
        hipDoubleComplex h_Mxt[gridsize_t];
        hipDoubleComplex h_Myt[gridsize_t];
        hipDoubleComplex h_Mzt[gridsize_t];
        
        hipDoubleComplex h_Ext[gridsize_t];
        hipDoubleComplex h_Eyt[gridsize_t];
        hipDoubleComplex h_Ezt[gridsize_t];
        
        hipDoubleComplex h_Hxt[gridsize_t];
        hipDoubleComplex h_Hyt[gridsize_t];
        hipDoubleComplex h_Hzt[gridsize_t];
        
        // Copy content of Device J,M,E,H arrays into Host J,M,E,H arrays
        hipMemcpy(h_Jxt, d_Jxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Jyt, d_Jyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Jzt, d_Jzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Mxt, d_Mxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Myt, d_Myt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Mzt, d_Mzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Ext, d_Ext, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Eyt, d_Eyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Ezt, d_Ezt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Hxt, d_Hxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hyt, d_Hyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hzt, d_Hzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // Free Device memory
        hipDeviceReset();
        
        // Pack CUDA arrays into cpp array for processing
        std::array<hipDoubleComplex*, 3> CJ;
        std::array<hipDoubleComplex*, 3> CM;
        
        std::array<hipDoubleComplex*, 3> CE;
        std::array<hipDoubleComplex*, 3> CH;
        
        // Fill the C++ std::array with C-style arrays
        CJ[0] = h_Jxt;
        CJ[1] = h_Jyt;
        CJ[2] = h_Jzt;
        
        CM[0] = h_Mxt;
        CM[1] = h_Myt;
        CM[2] = h_Mzt;
        
        CE[0] = h_Ext;
        CE[1] = h_Eyt;
        CE[2] = h_Ezt;
        
        CH[0] = h_Hxt;
        CH[1] = h_Hyt;
        CH[2] = h_Hzt;
        
        // Convert the CUDA style arrays to format compatible with CPU functions
        std::vector<std::array<std::complex<double>, 3>> Jt = ghandler.CUDAToCpp_C(CJ, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Mt = ghandler.CUDAToCpp_C(CM, gridsize_t);
        
        std::vector<std::array<std::complex<double>, 3>> Et = ghandler.CUDAToCpp_C(CE, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Ht = ghandler.CUDAToCpp_C(CH, gridsize_t);

        std::string Jt_file = "Jt";
        std::string Mt_file = "Mt";
        
        std::string Et_file = "Et";
        std::string Ht_file = "Ht";
        
        // Write using standard CPU DataHandler object
        ghandler.dh.writeOutC(Jt, Jt_file);
        ghandler.dh.writeOutC(Mt, Mt_file);
        
        ghandler.dh.writeOutC(Et, Et_file);
        ghandler.dh.writeOutC(Ht, Ht_file);
    }
    
    else if (toPrint == 3)
    {
        // Allocate memory for Pr, Er and Hr arrays on Device
        double *d_Prxt; hipMalloc( (void**)&d_Prxt, gridsize_t * sizeof(double) );
        double *d_Pryt; hipMalloc( (void**)&d_Pryt, gridsize_t * sizeof(double) );
        double *d_Przt; hipMalloc( (void**)&d_Przt, gridsize_t * sizeof(double) );
        
        hipDoubleComplex *d_Ext; hipMalloc( (void**)&d_Ext, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Eyt; hipMalloc( (void**)&d_Eyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Ezt; hipMalloc( (void**)&d_Ezt, gridsize_t * sizeof(hipDoubleComplex) );
        
        hipDoubleComplex *d_Hxt; hipMalloc( (void**)&d_Hxt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hyt; hipMalloc( (void**)&d_Hyt, gridsize_t * sizeof(hipDoubleComplex) );
        hipDoubleComplex *d_Hzt; hipMalloc( (void**)&d_Hzt, gridsize_t * sizeof(hipDoubleComplex) );
        
        // Call to KERNEL 3
        GpropagateBeam_3<<<nrb, nrt>>>(d_xs, d_ys, d_zs, 
                                   d_A, d_xt, d_yt, d_zt,
                                   d_nxt, d_nyt, d_nzt,
                                   d_Jx, d_Jy, d_Jz,
                                   d_Mx, d_My, d_Mz,
                                   d_Prxt, d_Pryt, d_Przt,
                                   d_Ext, d_Eyt, d_Ezt,
                                   d_Hxt, d_Hyt, d_Hzt);
        hipDeviceSynchronize();
        
        // Allocate, on stackframe, Host arrays for Pr, Er and Hr
        double h_Prxt[gridsize_t];
        double h_Pryt[gridsize_t];
        double h_Przt[gridsize_t];
        
        hipDoubleComplex h_Ext[gridsize_t];
        hipDoubleComplex h_Eyt[gridsize_t];
        hipDoubleComplex h_Ezt[gridsize_t];
        
        hipDoubleComplex h_Hxt[gridsize_t];
        hipDoubleComplex h_Hyt[gridsize_t];
        hipDoubleComplex h_Hzt[gridsize_t];
        
        // Copy content of Device Pr,Er,Hr arrays into Host Pr,Er,Hr arrays
        hipMemcpy(h_Prxt, d_Prxt, gridsize_t * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_Pryt, d_Pryt, gridsize_t * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_Przt, d_Przt, gridsize_t * sizeof(double), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Ext, d_Ext, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Eyt, d_Eyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Ezt, d_Ezt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        hipMemcpy(h_Hxt, d_Hxt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hyt, d_Hyt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        hipMemcpy(h_Hzt, d_Hzt, gridsize_t * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        
        // Free Device memory
        hipDeviceReset();
        
        // Pack CUDA arrays into cpp array for processing
        std::array<double*, 3> CP;
        std::array<hipDoubleComplex*, 3> CE;
        std::array<hipDoubleComplex*, 3> CH;
        
        // Fill the C++ std::array with C-style arrays
        CP[0] = h_Prxt;
        CP[1] = h_Pryt;
        CP[2] = h_Przt;
        
        CE[0] = h_Ext;
        CE[1] = h_Eyt;
        CE[2] = h_Ezt;
        
        CH[0] = h_Hxt;
        CH[1] = h_Hyt;
        CH[2] = h_Hzt;
        
        // Convert the CUDA style arrays to format compatible with CPU functions
        std::vector<std::array<double, 3>> Pr = ghandler.CUDAToCpp_R(CP, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Et = ghandler.CUDAToCpp_C(CE, gridsize_t);
        std::vector<std::array<std::complex<double>, 3>> Ht = ghandler.CUDAToCpp_C(CH, gridsize_t);
        
        std::string Pr_file = "Pr";
        std::string Et_file = "Et";
        std::string Ht_file = "Ht";
        
        // Write using standard CPU DataHandler object
        ghandler.dh.writeOutR(Pr, Pr_file);
        ghandler.dh.writeOutC(Et, Et_file);
        ghandler.dh.writeOutC(Ht, Ht_file);
    }
    return 0;
}
 
