#include "hip/hip_runtime.h"
#include "InterfaceCUDA.h"

/*! \file KernelsRTf.cu
    \brief Kernels for CUDA RT calculations.
    
    Contains kernels for RT calculations. Multiple kernels are defined, each one optimized for a certain calculation.
*/

// Declare constant memory for Device
__constant__ float conrt[CSIZERT]; // a, b, c, t0, epsilon
__constant__ float mat[16]; //
__constant__ int nTot;
__constant__ int cflip;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**
 * Check CUDA call.
 *
 * Wrapper for finding errors in CUDA API calls.
 *
 * @param code The errorcode returned from failed API call.
 * @param file The file in which failure occured.
 * @param line The line in file in which error occured.
 * @param abort Exit code upon error.
 */
__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * Debug real array.
 *
 * Print real valued array of size 3.
 *      Useful for debugging.

 * @param arr Array of 3 float.
 */
__host__ __device__ void _debugArrayf(float arr[3])
{
    printf("%f, %f, %f\n", arr[0], arr[1], arr[2]);
}

/**
  Calculate common factor 1.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  */
__device__ __inline__ float common1(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*(xr + t*dxr)/(conrt[0]*conrt[0]) + (yr + t*dyr)*(yr + t*dyr)/(conrt[1]*conrt[1]);
}

/**
  Calculate common factor 2.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  */
__device__ __inline__ float common2(float t, float xr, float yr, float dxr, float dyr)
{
    return (xr + t*dxr)*2*dxr/(conrt[0]*conrt[0]) + (yr + t*dyr)*2*dyr/(conrt[1]*conrt[1]);
}

/**
  Calculate ray-paraboloid intersection.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  @param dzr z co-ordinate of ray direction
  */
__device__ __inline__ float gp(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - common1(t, xr, yr, dxr, dyr)) /
                (dzr - common2(t, xr, yr, dxr, dyr));
}


/**
  Calculate ray-hyperboloid intersection.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  @param dzr z co-ordinate of ray direction
  */
__device__ __inline__ float gh(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - conrt[2]*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) /
                (dzr - conrt[2]/(2*sqrt(common1(t, xr, yr, dxr, dyr) + 1)) *
                common2(t, xr, yr, dxr, dyr));
}


/**
  Calculate ray-ellipsoid intersection.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  @param dzr z co-ordinate of ray direction
  */
__device__ __inline__ float ge(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr - conrt[2]*sqrt(1 - common1(t, xr, yr, dxr, dyr))) /
                (dzr + conrt[2]/(2*sqrt(1 - common1(t, xr, yr, dxr, dyr))) *
                common2(t, xr, yr, dxr, dyr));
}


/**
  Calculate ray-plane intersection.

  @param t Scaling factor.
  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param dxr x co-ordinate of ray direction
  @param dyr y co-ordinate of ray direction
  @param dzr z co-ordinate of ray direction
  */
__device__ __inline__ float gpl(float t, float xr, float yr, float zr, float dxr, float dyr, float dzr)
{
    return t - (zr + t*dzr) / dzr;
}


/**
  Calculate paraboloid normals.

  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param out Array of 3 float.
  */
__device__ __inline__ void np(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = -2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


/**
  Calculate hyperboloid normals.

  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param out Array of 3 float.
  */
__device__ __inline__ void nh(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = -2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = -2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = 2 * zr / (conrt[2]*conrt[2]) * cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}

/**
  Calculate ellipsoid normals.

  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param out Array of 3 float.
  */
__device__ __inline__ void ne(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = 2 * xr / (conrt[0]*conrt[0]) * cflip;
    out[1] = 2 * yr / (conrt[1]*conrt[1]) * cflip;
    out[2] = 2 * zr / (conrt[2]*conrt[2]) * cflip;

    float norm = sqrt(out[0]*out[0] + out[1]*out[1] + out[2]*out[2]);

    out[0] = out[0] / norm;
    out[1] = out[1] / norm;
    out[2] = out[2] / norm;
}


/**
  Calculate plane normals.

  @param xr x co-ordinate of ray.
  @param yr y co-ordinate of ray.
  @param zr z co-ordinate of ray.
  @param out Array of 3 float.
  */
__device__ __inline__ void npl(float xr, float yr, float zr, float (&out)[3])
{
    out[0] = 0;
    out[1] = 0;
    out[2] = cflip;
}

/**
  Matrix-vector multiplication.

  Uses mat from constant memory.

  @param cv1 Array of 3 float.
  @param out Array of 3 float.
  @param vec Whether to rotate as a vector or as a point.
  */
__device__ __inline__ void matVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2];
        }
    }

    else
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n*4] * cv1[0] + mat[1+n*4] * cv1[1] + mat[2+n*4] * cv1[2] + mat[3+n*4];
        }
    }
}

/**
  Matrix-vector multiplication.

  Uses mat from constant memory.

  @param cv1 Array of 3 float.
  @param out Array of 3 float.
  @param vec Whether to rotate as a vector or as a point.
  */
__device__ __inline__ void invmatVec4(float (&cv1)[3], float (&out)[3], bool vec = false)
{
    if (vec)
    {
        for(int n=0; n<3; n++)
        {
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2];
        }
    }

    else
    {
        float temp;
        for(int n=0; n<3; n++)
        {
            temp = -mat[n]*mat[3] - mat[n+4]*mat[7] - mat[n+8]*mat[11];
            out[n] = mat[n] * cv1[0] + mat[n+4] * cv1[1] + mat[n+8] * cv1[2] + temp;
        }
    }
}

/**
  Transform rays to surface restframe.

  @param x Array of ray x co-ordinates.
  @param y Array of ray y co-ordinates.
  @param z Array of ray z co-ordinates.
  @param dx Array of ray x directions.
  @param dy Array of ray y directions.
  @param dz Array of ray z directions.
  @param i Index of co-ordinate.
  @param inv Whether to apply inverse transformation.
  */
__device__ __inline__ void transfRays(float *x, float *y, float *z,
                                  float *dx, float *dy, float *dz,
                                  int i, bool inv = false)
{
    bool vec = true;
    float inp[3], out[3];

    //if (i == 300) {printf("%f\n", mat[0]);}

    inp[0] = x[i];
    inp[1] = y[i];
    inp[2] = z[i];

    if (inv) {invmatVec4(inp, out);}
    else {matVec4(inp, out);}

    x[i] = out[0];
    y[i] = out[1];
    z[i] = out[2];

    inp[0] = dx[i];
    inp[1] = dy[i];
    inp[2] = dz[i];

    if (inv) {invmatVec4(inp, out, vec);}
    else {matVec4(inp, out, vec);}

    dx[i] = out[0];
    dy[i] = out[1];
    dz[i] = out[2];

}

/**
 * Initialize CUDA.
 *
 * Instantiate program and populate constant memory.
 *
 * @param ctp reflparamsf object containing target surface parameters.
 * @param epsilon Precision of NR method.
 * @param t0 Starting guess for NR method.
 * @param _nTot Total number of rays in beam.
 * @param nBlocks Number of blocks per grid.
 * @param nThreads Number of threads per block.
 *
 * @return BT Array of two dim3 objects.
 */
__host__ std::array<dim3, 2> _initCUDA(reflparamsf ctp, float epsilon, float t0,
                                      int _nTot, int nBlocks, int nThreads)
{
    // Calculate nr of blocks per grid and nr of threads per block
    dim3 nrb(nBlocks); dim3 nrt(nThreads);

    // Pack constant array
    hipFloatComplex _conrt[CSIZERT] = {ctp.coeffs[0], ctp.coeffs[1],
                                  ctp.coeffs[2], t0, epsilon};

    int iflip = 1;
    if (ctp.flip) {iflip = -1;}

    // Copy constant array to Device constant memory
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(conrt), &_conrt, CSIZERT * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(mat), ctp.transf, 16 * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(nTot), &_nTot, sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cflip), &iflip, sizeof(int)) );

    std::array<dim3, 2> BT;
    BT[0] = nrb;
    BT[1] = nrt;

    return BT;
}

/**
  Optimize ray-paraboloid distance.

  Uses a Newton Rhapson (NR) method to find the point of ray-surface intersection.

  @param xs Array of ray x co-ordinates.
  @param ys Array of ray y co-ordinates.
  @param zs Array of ray z co-ordinates.
  @param dxs Array of ray x directions.
  @param dys Array of ray y directions.
  @param dzs Array of ray z directions.
  @param xt Array of ray x co-ordinates, to be filled.
  @param yt Array of ray y co-ordinates, to be filled.
  @param zt Array of ray z co-ordinates, to be filled.
  @param dxt Array of ray x directions, to be filled.
  @param dyt Array of ray y directions, to be filled.
  @param dzt Array of ray z directions, to be filled.
  */
__global__ void propagateRaysToP(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gp(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }

        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        np(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

/**
  Optimize ray-hyperboloid distance.

  Uses a Newton Rhapson (NR) method to find the point of ray-surface intersection.

  @param xs Array of ray x co-ordinates.
  @param ys Array of ray y co-ordinates.
  @param zs Array of ray z co-ordinates.
  @param dxs Array of ray x directions.
  @param dys Array of ray y directions.
  @param dzs Array of ray z directions.
  @param xt Array of ray x co-ordinates, to be filled.
  @param yt Array of ray y co-ordinates, to be filled.
  @param zt Array of ray z co-ordinates, to be filled.
  @param dxt Array of ray x directions, to be filled.
  @param dyt Array of ray y directions, to be filled.
  @param dzt Array of ray z directions, to be filled.
  */
__global__ void propagateRaysToH(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gh(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        nh(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

/**
  Optimize ray-ellipsoid distance.

  Uses a Newton Rhapson (NR) method to find the point of ray-surface intersection.

  @param xs Array of ray x co-ordinates.
  @param ys Array of ray y co-ordinates.
  @param zs Array of ray z co-ordinates.
  @param dxs Array of ray x directions.
  @param dys Array of ray y directions.
  @param dzs Array of ray z directions.
  @param xt Array of ray x co-ordinates, to be filled.
  @param yt Array of ray y co-ordinates, to be filled.
  @param zt Array of ray z co-ordinates, to be filled.
  @param dxt Array of ray x directions, to be filled.
  @param dyt Array of ray y directions, to be filled.
  @param dzt Array of ray z directions, to be filled.
  */
__global__ void propagateRaysToE(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < nTot)
    {
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];

        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = ge(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        ne(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

/**
  Optimize ray-plane distance.

  Uses a Newton Rhapson (NR) method to find the point of ray-surface intersection.

  @param xs Array of ray x co-ordinates.
  @param ys Array of ray y co-ordinates.
  @param zs Array of ray z co-ordinates.
  @param dxs Array of ray x directions.
  @param dys Array of ray y directions.
  @param dzs Array of ray z directions.
  @param xt Array of ray x co-ordinates, to be filled.
  @param yt Array of ray y co-ordinates, to be filled.
  @param zt Array of ray z co-ordinates, to be filled.
  @param dxt Array of ray x directions, to be filled.
  @param dyt Array of ray y directions, to be filled.
  @param dzt Array of ray z directions, to be filled.
  */
__global__ void propagateRaysToPl(float *xs, float *ys, float *zs,
                                float *dxs, float *dys, float *dzs,
                                float *xt, float *yt, float *zt,
                                float *dxt, float *dyt, float *dzt)
{
    float norms[3];
    bool inv = true;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx < nTot)
    {
      //if (idx == 0) {printf("%f\n", conrt[2]);}
        transfRays(xs, ys, zs, dxs, dys, dzs, idx, inv);

        float _t = conrt[3];
        float t1 = 1e99;

        float check = fabs(t1 - _t);

        float x = xs[idx];
        float y = ys[idx];
        float z = zs[idx];
        //printf("%f\n", x);
        float dx = dxs[idx];
        float dy = dys[idx];
        float dz = dzs[idx];

        while (check > conrt[4])
        {
            t1 = gpl(_t, x, y, z, dx, dy, dz);

            check = fabs(t1 - _t);

            _t = t1;
        }
        //printf("%.14f\n", check);

        xt[idx] = x + _t*dx;
        yt[idx] = y + _t*dy;
        zt[idx] = z + _t*dz;

        npl(xt[idx], yt[idx], zt[idx], norms);
        check = (dx*norms[0] + dy*norms[1] + dz*norms[2]);

        dxt[idx] = dx - 2*check*norms[0];
        dyt[idx] = dy - 2*check*norms[1];
        dzt[idx] = dz - 2*check*norms[2];

        transfRays(xs, ys, zs, dxs, dys, dzs, idx);
        transfRays(xt, yt, zt, dxt, dyt, dzt, idx);
    }
}

/**
  Call ray-trace Kernel.

  Calculate a new frame of rays. Several kernels can be called, depending on surface type.

  @param ctp reflparamsf object containing target surface parameters.
  @param fr_in Pointer to input cframef object.
  @param fr_out Pointer to output cframef object.
  @param epsilon Precision for NR method.
  @param t0 Starting guess for NR method.
  @param nBlocks Number of blocks in GPU grid.
  @param nThreads Number of threads in block.
  
  @see reflparamsf
  @see cframef
  */
void callRTKernel(reflparamsf ctp, cframef *fr_in,
                            cframef *fr_out, float epsilon, float t0,
                            int nBlocks, int nThreads)
{
    std::array<dim3, 2> BT;
    BT = _initCUDA(ctp, epsilon, t0, fr_in->size, nBlocks, nThreads);

    float *d_xs, *d_ys, *d_zs;
    gpuErrchk( hipMalloc((void**)&d_xs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_ys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_xs, fr_in->x, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_ys, fr_in->y, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_zs, fr_in->z, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_dxs, *d_dys, *d_dzs;
    gpuErrchk( hipMalloc((void**)&d_dxs, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dys, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzs, fr_in->size * sizeof(float)) );

    gpuErrchk( hipMemcpy(d_dxs, fr_in->dx, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dys, fr_in->dy, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dzs, fr_in->dz, fr_in->size * sizeof(float), hipMemcpyHostToDevice) );

    float *d_xt, *d_yt, *d_zt;
    gpuErrchk( hipMalloc((void**)&d_xt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_yt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_zt, fr_in->size * sizeof(float)) );

    float *d_dxt, *d_dyt, *d_dzt;
    gpuErrchk( hipMalloc((void**)&d_dxt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dyt, fr_in->size * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_dzt, fr_in->size * sizeof(float)) );

    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;

    printf("Calculating ray-trace...\n");
    begin = std::chrono::steady_clock::now();

    if (ctp.type == 0)
    {
        propagateRaysToP<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        //printf("made it\n");
        gpuErrchk( hipDeviceSynchronize() );

    }

    else if (ctp.type == 1)
    {
        propagateRaysToH<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    else if (ctp.type == 2)
    {
        propagateRaysToE<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    else if (ctp.type == 3)
    {
        propagateRaysToPl<<<BT[0], BT[1]>>>(d_xs, d_ys, d_zs, d_dxs, d_dys, d_dzs,
                                          d_xt, d_yt, d_zt, d_dxt, d_dyt, d_dzt);
        gpuErrchk( hipDeviceSynchronize() );
    }

    end = std::chrono::steady_clock::now();
    std::cout << "Elapsed time : "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()
              << " [us]\n" << std::endl;

    gpuErrchk( hipMemcpy(fr_out->x, d_xt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->y, d_yt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->z, d_zt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(fr_out->dx, d_dxt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dy, d_dyt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(fr_out->dz, d_dzt, fr_in->size * sizeof(float), hipMemcpyDeviceToHost) );
    
    gpuErrchk( hipDeviceReset() );
}
